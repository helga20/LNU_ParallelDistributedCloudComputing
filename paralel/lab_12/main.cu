
#include <hip/hip_runtime.h>
#include <ios>
#include <iostream>
#include <chrono>
#include <numeric>
#include <string>
#include <vector>
// #include <iomanip>
using std::cout, std::vector,
	  std::chrono::high_resolution_clock, std::chrono::microseconds, std::chrono::duration_cast;


double round(double x)
{
	int k = std::pow(10, 2);
	return std::ceil(x * k) / k;
}

class TablePrint
{
	vector<int> spaces;
	int count = 0;
public:
	TablePrint(std::initializer_list<int> spaces_list)
	{
		this->spaces = spaces_list;
		this->count = std::accumulate(spaces.begin(), spaces.end(), 1 + spaces.size());
	}
	void print_line()
	{
		for(int i = 0; i < count; i++)
			cout << "-";
		cout << '\n';
	}
	template< typename ... ARGS>
	void print(ARGS&& ... args)
	{
		int i = 0;
		cout << "|";
		((cout << args << "|"), ...);
		cout << '\n';
	}
};



__global__ void sum_of_element_in_row(int* matrix, int rows, int columns)
{
	int block_id = threadIdx.x + blockIdx.x * blockDim.x;

	int* result;
	for(int i = 0; i < rows; i++)
	{
		int sum = 0;
		for(int j = 0; j < columns; j++)
			sum += matrix[i * columns + j + block_id];
		// result[i] = sum;
	}
}


void random_matrix(int* matrix, int rows_count, int columns_count)
{
	for(int i = 0; i < rows_count; i++)
		for(int j = 0; j < columns_count; j++)
			matrix[i * columns_count + j] = rand() % 100;
}
int get_time(int rows_count, int columns_count, int threads_count = 1)
{
	int* matrix;
	hipHostMalloc((void **) &matrix, sizeof(int) * rows_count * columns_count, hipHostMallocDefault);


	random_matrix(matrix, rows_count, columns_count);


	auto start = high_resolution_clock::now();

	sum_of_element_in_row<<<1, threads_count>>>(matrix, rows_count, columns_count);
	hipDeviceSynchronize();

	auto stop = high_resolution_clock::now();

	hipFree(matrix);
	return duration_cast<microseconds>(stop - start).count();
}
int main()
{
	TablePrint table({7, 5, 9, 11, 13});
	table.print("threads", "size", "sync time", "thread time", "acceleration");

	for(int thread_count = 2; thread_count <= 256; thread_count *= 2)
		for(int i = 10; i <= 10000; i *= 10)
		{
			int sync = get_time(i, i, 1);
			int thread = get_time(i, i, thread_count);
			double acceleration = round(double(sync) / std::max(1, thread));
			table.print(thread_count, i, sync, thread, acceleration);
		}

}
