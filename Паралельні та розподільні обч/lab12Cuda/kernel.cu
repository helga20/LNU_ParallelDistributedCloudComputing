#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void update(float* life, int* numParticles)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (life[idx] > 0) {
        (*numParticles)++;
    }
}

int main()
{
    int numParticles = 0;
    int* numParticles_d = 0;
    hipMalloc((void**)&numParticles_d, sizeof(int));
    update << <MAX_PARTICLES / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (life, numParticles_d);
    hipMemcpy(&numParticles, numParticles_d, sizeof(int), hipMemcpyDeviceToHost);
    printf("%s %i", "numParticles: ", numParticles);

    return 0;
}


